//Neel V Zadafiya (1115533)
//Condition_1.cu
//Runtime: Visual Studio (Using NVCC)
//Assignment 3 Part 2 - Matrix Multiplication using CUDA
//GPU: RTX 2080 Ti 11 GB

//Libraries for cuda runtime
#include "hip/hip_runtime.h"


//Standard C libraries
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//Function for matrix multiplication
//C(m,w) = A(m,n) X B(n,w)
__global__ void matMul(int *c, int *a, int *b, int *m, int *n, int *w)
{
	//Get unique index of thread
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	
	//Get row and column value from flatten matrix
	int row = x / *w;
	int col = x % *w;
	
	//Calculate offset for result matrix
    int offset = row * *w + col;
	
	//Initialize sum to zero
	int sum = 0;
	
	//Loop to calculate actual multiplication
	for(int i =0; i<*n; i++)
	{
		//C[i][j] += A[i][k] * B[k][j] ; Innermost loop of standard matrix multiplication algorithm
		sum += a[(row * (*n))+i] * b[col+(i* (*w))];
	}
	
	//Store the sum in respective cell
	c[offset] = sum;
}

//Main function
int main()
{
	//Initialize start time
	clock_t startTime = clock();
	
	//C(m,w) = A(m,n) X B(n,w)
	//Initialize conditions
	int m = 500;
	int n = 500;
	int w = 400;
	int N = 100;
	
	//Initialize pointers for host array
    int *a;
	int *b;
	int *c;
	
	//Allocate host memory for above pointers
	a = (int *)malloc(m*n*sizeof(int));
	b = (int *)malloc(n*w*sizeof(int));
	c = (int *)malloc(m*w*sizeof(int));
	
	//Initialize random number generator
	srand(time(0));
	
	//Generate random numbers for B
	for(int i=0;i<n*w;i++)
	{
		b[i] = rand() % 10;
	}
	
	//Initialize pointers for device array and dimentions
	int *d_a, *d_b, *d_c;
	int *d_m, *d_n, *d_w;
	
	//Allocate memory to B in device
	hipMalloc((void **)&d_b, n * w * sizeof(int));
	
	//Allocate memory for dimentions in device
	hipMalloc((void **)&d_m, sizeof(int));
	hipMalloc((void **)&d_n, sizeof(int));
	hipMalloc((void **)&d_w, sizeof(int));
	
	//Copy values of dimentions from host to device
	hipMemcpy(d_m, &m, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_w, &w, sizeof(int), hipMemcpyHostToDevice);
	
	//Copy values of B from host to device
	hipMemcpy(d_b, b, n * w * sizeof(int), hipMemcpyHostToDevice);
	
	//Core loop to iterate N
	for(int z = 0; z<N; z++)
	{
	
		//Generate random numbers for A
		for(int i=0;i<m*n;i++)
		{
			a[i] = rand() % 10;
		}
		
		//Allocate memory to A and C in device
		hipMalloc((void **)&d_a, m * n * sizeof(int));
		hipMalloc((void **)&d_c, m * w * sizeof(int));
		
		//Copy values of A from host to device
		hipMemcpy(d_a, a, m * n * sizeof(int), hipMemcpyHostToDevice);
		
		//Call kernel function
		matMul<<<200,1000>>>(d_c, d_a, d_b, d_m, d_n, d_w);
		
		//Transfer results back to host memory
		hipMemcpy(c, d_c, m * w * sizeof(int), hipMemcpyDeviceToHost);
		
		//Free memory of A and C from device
		hipFree(d_a);
		hipFree(d_c);
		
		//Un-comment the code given below to print output of matrices on every iteration
		
		/*printf("===========================\n");
		printf("Iteration : %d\n\n",z);
		
		printf("Values of A:\n");
		for(int i=0;i<m;i++)
		{
			for(int j=0;j<n;j++)
			{
				printf("%d ",a[i*n+j]);
			}
			printf("\n");
		}
		
		printf("\nValues of B:\n");
		
		for(int i=0;i<n;i++)
		{
			for(int j=0;j<w;j++)
			{
				printf("%d ",b[i*w+j]);
			}
			printf("\n");
		}
		
		printf("\nValues of C:\n");
		
		for(int i=0;i<m;i++)
		{
			for(int j=0;j<w;j++)
			{
				printf("%d ",c[i*w+j]);
			}
			printf("\n");
		}
		printf("\n");*/
		
	}
	
	//Free memory of B and dimentions from device
	hipFree(d_b);
	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_w);

	//Initialize end time
	clock_t endTime = clock();
	
	//Print time taken by the program
	printf("Elapsed: %f seconds\n", (double)(endTime - startTime) / CLOCKS_PER_SEC);
	
	//Return 0 to finisdh the main function
    return 0;
}